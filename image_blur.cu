
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>


__global__ void imageblur( int* inputImage, int* outputImage, int filterSize,  double* filter, int imageRow, int imageCol){
  
  int pixelx = blockIdx.x * blockDim.x + threadIdx.x;
  int pixely = blockIdx.y * blockDim.y + threadIdx.y;
  double blur_value = 0.0;  
  
  if (pixelx >= imageCol || pixely >= imageRow) {
      return;
  }

  //multiply with blur kernel
  for (int finalx = 0; finalx < filterSize; finalx++) {
    for (int finaly = 0; finaly < filterSize; finaly++) {
      int imagex = pixelx + finalx - filterSize / 2 ;
      int imagey = pixely + finaly - filterSize / 2;
      int imagePixel;
      if(imagex < 0 || imagex >= imageCol || imagey < 0 || imagey >= imageRow){
        imagePixel = 0;
      } else {
        imagePixel = inputImage[imagey*imageCol+imagex];
      }
      blur_value += (filter[finaly*filterSize+finalx] * imagePixel);
    }
  }

  outputImage[pixely*imageCol+pixelx] = (int)(blur_value/15.0);
}

int main(int argc, char const *argv[]) {

  int imagex = 3, imagey = 3;
  
  int numberOfPixels = imagex*imagey*sizeof(int);
  
  int *d_image = 0; int *d_blurImage = 0; double *d_filter = 0; //device
  int *h_image = 0; int *h_blurImage = 0; double *h_filter = 0; //host
  
  //malloc memory device and host
  h_image = (int*)malloc(numberOfPixels);
  hipMalloc((void**)&d_image, numberOfPixels);
  h_blurImage = (int*)malloc(numberOfPixels);
  hipMalloc((void**)&d_blurImage, numberOfPixels);
  h_filter = (double*)malloc(9*sizeof(double));
  hipMalloc((void**)&d_filter, 9*sizeof(double));

  if(h_image == 0 || d_image == 0 || h_blurImage == 0 || d_blurImage == 0){
    printf("Could not allocate memory");
    return 1;
  }
  
  //Initialise Filter
  h_filter[0] = 1.0; h_filter[1] = 2.0; h_filter[2] = 1.0;
  h_filter[3] = 2.0; h_filter[4] = 3.0; h_filter[5] = 2.0;
  h_filter[6] = 1.0; h_filter[7] = 2.0; h_filter[8] = 1.0;

  // Randomly Initialize Image
  srand(time(NULL));
  for(int i = 0; i < (imagex*imagey); i++){
    h_image[i] = (rand() % 256);
  }
  
  //Copy host memory to device
  hipMemcpy( d_image, h_image, numberOfPixels, hipMemcpyHostToDevice);
  hipMemcpy( d_filter, h_filter, 9*sizeof(double), hipMemcpyHostToDevice);
  
  const dim3 blockSize(4,4,1);
  const dim3 gridSize(imagex/blockSize.x+1,imagey/blockSize.y+1,1);
  
  //Call
  imageblur<<<gridSize, blockSize>>>(d_image, d_blurImage, 3, d_filter, imagey, imagex);
  
  //copy blurred image to host
  hipMemcpy(h_blurImage, d_blurImage, numberOfPixels, hipMemcpyDeviceToHost);

  printf("Image : \n");
  for(int i = 0; i < imagex; i++){
    for(int j = 0; j < imagey; j++){
      printf("%d ", h_image[i*imagex + j]);
    }
    printf("\n");
  }
  printf("Blur Image: \n");

  for(int i = 0; i < imagex; i++){
    for(int j = 0; j < imagey; j++){
      printf("%d ", h_blurImage[i*imagex + j]);
    }
    printf("\n");
  }
  
  //Clean Memory
  free(h_image); free(h_blurImage); free(h_filter);
  hipFree(d_image); hipFree(d_blurImage); hipFree(d_filter);

  return 0;
}

